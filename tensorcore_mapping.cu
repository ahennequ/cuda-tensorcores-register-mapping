
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <type_traits>
#include <mma.h>
using namespace nvcuda;

// Check tensor core's warp register layout
// nvcc -arch=sm_75 tensorcore_mapping.cu -o mapping
// ./mapping

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

template<typename scalar_t>
__device__ int getWarpRow(int i) {
  int tid = threadIdx.x % 32;
  #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 700)
    if (std::is_same<scalar_t, half>::value) {
      return (tid & 3) + ((tid & 4) << 1) + ((tid & 16) >> 2);
    } else {
      return (tid & 16) / 4 + 2 * (tid & 4) + (tid & 1) + (i & 2);
    }
  #else
    return (i & 2) * 4 + tid / 4;
  #endif
}

template<typename scalar_t>
__device__ int getWarpCol(int i) {
  int tid = threadIdx.x % 32;
  #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 700)
    if (std::is_same<scalar_t, half>::value) {
      return (i & 7) + (tid & 8);
    } else {
      return (tid & 10) + (i & 5);
    }
  #else
    return (tid % 4) * 2 + i % 2 + (i & 4) * 2;
  #endif
}

template<int N, int M, typename T>
__global__ void wmma_example(T *elem, T* thread, T* row, T* col) {
  #if defined(__CUDA_ARCH__)
    if (threadIdx.x == 0) printf("cuda %d\n", __CUDA_ARCH__);
  #endif

   wmma::fragment<wmma::accumulator, N, M, 16, T> acc_frag;

   wmma::fill_fragment(acc_frag, 0.0f);
   for (int i=0 ; i<acc_frag.num_elements; i++) {
    acc_frag.x[i] = i;
   }
   wmma::store_matrix_sync(elem, acc_frag, M, wmma::mem_row_major);

   wmma::fill_fragment(acc_frag, 0.0f);
   for (int i=0 ; i<acc_frag.num_elements; i++) {
    acc_frag.x[i] = threadIdx.x;
   }
   wmma::store_matrix_sync(thread, acc_frag, M, wmma::mem_row_major);

   // row:
   wmma::fill_fragment(acc_frag, 0.0f);
   for (int i=0 ; i<acc_frag.num_elements; i++) {
    acc_frag.x[i] = getWarpRow<T>(i);
   }
   wmma::store_matrix_sync(row, acc_frag, M, wmma::mem_row_major);

   // col:
   wmma::fill_fragment(acc_frag, 0.0f);
   for (int i=0 ; i<acc_frag.num_elements; i++) {
    acc_frag.x[i] = getWarpCol<T>(i);
   }
   wmma::store_matrix_sync(col, acc_frag, M, wmma::mem_row_major);
}

#include <vector>
struct MaskShift {
  int var;
  int mask;
  int shift;

  void prettyPrint() {
    const char* varname;
    if (var == 0) {
      varname = "i";
    } else {
      varname = "tid";
    }
    if (shift < 0) {
      printf("((%s & %d) >> %d)", varname, mask, -shift);
    } else if (shift > 0) {
      printf("((%s & %d) << %d)", varname, mask, shift);
    } else {
      printf("(%s & %d)", varname, mask);
    }
  }
};

template<int N>
void appendVar(int var, std::vector<MaskShift>& formula, int* invariant) {
  int inv_mask = 0;
  for (int i = 0; i < N; i++) {
    inv_mask |= invariant[i];
  }

  for (int bit = 0; bit < 8; bit++) {
    if (((inv_mask >> bit) & 1) == 0) continue;
    for (int bit2 = 0; bit2 < 4; bit2++) {
      bool correlated = true;
      for (int i = 0; i < N; i++) {
        if (((i >> bit2) & 1) != ((invariant[i] >> bit) & 1)) {
          correlated = false;
          break;
        }
      }
      if (correlated) {
        bool added = false;
        int shift = bit2 - bit;
        for (auto& ms : formula) {
          if (ms.var == var && ms.shift == shift) {
            ms.mask |= 1 << bit;
            added = true;
            break;
          }
        }
        if (!added) {
          formula.push_back({var, 1 << bit, shift});
        }
      }
    }
  }
}

template<int N, int M, typename T>
void find_formulae(T* elem, T* thread) {
  std::vector<MaskShift> row_formula;
  std::vector<MaskShift> col_formula;

  int row_invariant_elem[N];
  int row_invariant_thread[N];
  for (int i=0; i<N ; i++) {
    row_invariant_elem[i] = -1;
    row_invariant_thread[i] = -1;
    for (int j=0; j<M; j++) {
      row_invariant_elem[i] &= (int)(float)elem[i*M+j];
      row_invariant_thread[i] &= (int)(float)thread[i*M+j];
    }
  }

  int col_invariant_elem[M];
  int col_invariant_thread[M];
  for (int i=0; i<M ; i++) {
    col_invariant_elem[i] = -1;
    col_invariant_thread[i] = -1;
    for (int j=0; j<N; j++) {
      col_invariant_elem[i] &= (int)(float)elem[j*M+i];
      col_invariant_thread[i] &= (int)(float)thread[j*M+i];
    }
  }

  appendVar<N>(0, row_formula, row_invariant_elem);
  appendVar<N>(1, row_formula, row_invariant_thread);

  printf("Row:\nreturn ");
  const char* pad = "";
  for (auto ms : row_formula) {
    printf("%s", pad);
    ms.prettyPrint();
    pad = " + ";
  }
  printf(";\n\n");

  appendVar<M>(0, col_formula, col_invariant_elem);
  appendVar<M>(1, col_formula, col_invariant_thread);

  printf("Col:\nreturn ");
  pad = "";
  for (auto ms : col_formula) {
    printf("%s", pad);
    ms.prettyPrint();
    pad = " + ";
  }
  printf(";\n\n");
}

template<int N, int M, typename T>
void print_matrix(T* mat) {
  for (int i=0; i<N ; i++) {
    for (int j=0; j<M; j++) {
      printf("%2d ", (int)(float) mat[i*M+j]);
    }
    printf("\n");
  }
  printf("\n");
}

template<int N, int M, typename T>
void print_matrices(T* mat, T* mat2) {
  for (int i=0; i<N ; i++) {
    for (int j=0; j<M; j++) {
      printf("%2d ", (int)(float) mat[i*M+j]);
    }
    printf("\t");
    for (int j=0; j<M; j++) {
      printf("%2d ", (int)(float) mat2[i*M+j]);
    }
    printf("\n");
  }
  printf("\n");
}

int main(int argc, char* argv[]) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("%s\n", prop.name);

  using F = float;
  static const int N = 32;
  static const int M = 8;

  F *elem;
  F *thread;
  F *row;
  F *col;

  F *elem_host;
  F *thread_host;
  F *row_host;
  F *col_host;

  // Use tensor cores

  cudaErrCheck(hipMalloc((void**)&elem, N * M * sizeof(F)));
  cudaErrCheck(hipMalloc((void**)&thread, N * M * sizeof(F)));
  cudaErrCheck(hipMalloc((void**)&row, N * M * sizeof(F)));
  cudaErrCheck(hipMalloc((void**)&col, N * M * sizeof(F)));

  elem_host = (F*)malloc(N * M * sizeof(F));
  thread_host = (F*)malloc(N * M * sizeof(F));
  row_host = (F*)malloc(N * M * sizeof(F));
  col_host = (F*)malloc(N * M * sizeof(F));
  
  // First: using WMMA
  dim3 gridDim(1);
  dim3 blockDim(32);
  
  printf("Running with wmma...\n");
  wmma_example<N, M> <<< gridDim, blockDim >>> (elem, thread, row, col);

  // Error checking
  printf("\nChecking results...\n");
  cudaErrCheck(hipMemcpy(elem_host, elem, N * M * sizeof(F), hipMemcpyDeviceToHost));
  cudaErrCheck(hipMemcpy(thread_host, thread, N * M * sizeof(F), hipMemcpyDeviceToHost));
  cudaErrCheck(hipMemcpy(row_host, row, N * M * sizeof(F), hipMemcpyDeviceToHost));
  cudaErrCheck(hipMemcpy(col_host, col, N * M * sizeof(F), hipMemcpyDeviceToHost));
  
  printf("Elem / ThreadIdx:\n");
  print_matrices<N, M>(elem_host, thread_host);

  find_formulae<N, M>(elem_host, thread_host);

  printf("Verify row / col:\n");
  print_matrices<N, M>(row_host, col_host);

  cudaErrCheck(hipFree(elem));
  cudaErrCheck(hipFree(thread));
  cudaErrCheck(hipFree(row));
  cudaErrCheck(hipFree(col));
  
  free(elem_host);
  free(thread_host);
  free(row_host);
  free(col_host);

  cudaErrCheck(hipDeviceReset());
  return 0;
}